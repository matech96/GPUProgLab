#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t maxWithCuda(int* data, unsigned int dataSize);
hipError_t compactWithCuda(int* data, int* keep_data, int* not_keep_data, int* zero_offset_data, int* max_data, unsigned int nloops, unsigned int dataSize);

__global__
void mapNotKernel(int* x, unsigned int dataSize)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id > dataSize) return;
    x[id] = x[id] == 0 ? 1 : 0;
}

__global__
void mapBitKernel(int* x, int n, unsigned int dataSize)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id > dataSize) return;
    x[id] = (x[id] >> n) & 1U;
}

__global__
void predNonnegReduceKernel(int* data, int* keep, unsigned int dataSize)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (keep[id] == 0) data[id] = -1;
    for (int s = dataSize / 2; s > 0; s >>= 1) {
        if (id < s) {
            data[id] = (data[id] > data[id + s]) ? data[id] : data[id + s];
        }
        __syncthreads();
    }
}

__global__
void reduceKernel(int* data, unsigned int dataSize)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    for (int s = dataSize / 2; s > 0; s >>= 1) {
        if (id < s) {
            data[id] = (data[id] > data[id + s]) ? data[id] : data[id + s];
        }
        __syncthreads();
    }
}

__global__
void exscanKernel(int* data, unsigned int dataSize)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id > 0) data[id] = data[id - 1];
    __syncthreads();
    if (id == 0) data[id] = 0;
    __syncthreads();

    for (int s = 1; s < dataSize; s *= 2) {
        int tmp = data[id];
        if (id + s < dataSize) data[id + s] += tmp;
        __syncthreads();
    }
    if (id == 0) data[id] = 0;
}

__global__
void compactMapKernel(int* new_data, int* data, int* zero_offset_data, unsigned int zeroOffsetMax, int* one_offset_data, int* keep_data, unsigned int dataSize)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id > dataSize) return; 
    if (keep_data[id] == 1) new_data[zeroOffsetMax + one_offset_data[id]] = data[id];
    else new_data[zero_offset_data[id]] = data[id];
}

int main()
{
    int data[8] = { 2, 36, 8, 11, 5, 20, 55, 1 };
    int max_data[8] = { 2, 36, 8, 11, 5, 20, 55, 1 };
    int keep_data[8] = { 0,1,1,0,0,1,1,1 };
    int not_keep_data[8] = { 0,1,1,0,0,1,1,1 };
    int zoffset_data[8] = { 0,1,1,0,0,1,1,1 };

    hipError_t cudaStatus = maxWithCuda(max_data, 8);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "maxWithCuda failed!");
        return 1;
    }

    int max = max_data[0];
    unsigned int bitnum = 0;
    while (max > 0) {
        max = max >> 1;
        ++bitnum;
    }

    // Do the operation on vectors in parallel.
    cudaStatus = compactWithCuda(data, keep_data, not_keep_data, zoffset_data, max_data, bitnum, 8);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "compactWithCuda failed!");
        return 1;
    }

    printf("{ 2, 36, 8, 11, 5, 20, 55, 1 } radix sorted is {%d,%d,%d,%d,%d,%d,%d,%d}\n", data[0], data[1], data[2], data[3], data[4], data[5], data[6], data[7]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t maxWithCuda(int* data, unsigned int dataSize)
{
    int* dev_data = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (three input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_data, dataSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_data, data, dataSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    reduceKernel << <1, dataSize >> > (dev_data, dataSize);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "reduceKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching reduceKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(data, dev_data, dataSize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_data);

    return cudaStatus;
}

hipError_t compactWithCuda(int* data, int* keep_data, int* not_keep_data, int* zero_offset_data, int* max_data, unsigned int nloops, unsigned int dataSize)
{
    int* keep_dev_data = 0;
    int* notkeep_dev_data = 0;
    int* zoffset_dev_data = 0;
    int* ooffset_dev_data = 0;
    int* max_dev_data = 0;
    int* dev_data = 0;
    int* dev_new_data = 0;
    hipError_t cudaStatus;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);

    for(int i=0;i<nloops;++i){
        
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&keep_dev_data, dataSize * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }
        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpy(keep_dev_data, data, dataSize * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        mapBitKernel << <1, dataSize >> > (keep_dev_data, i, dataSize);
        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "mapBitKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching mapBitKernel!\n", cudaStatus);
            goto Error;
        }
        cudaStatus = hipMemcpy(keep_data, keep_dev_data, dataSize * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }


        cudaStatus = hipMalloc((void**)&ooffset_dev_data, dataSize * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }
        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpy(ooffset_dev_data, keep_data, dataSize * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        // Launch a kernel on the GPU with one thread for each element.
        exscanKernel << <1, dataSize >> > (ooffset_dev_data, dataSize);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "exscanKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching exscanKernel!\n", cudaStatus);
            goto Error;
        }


        // Allocate GPU buffers for three vectors (three input, one output)
        cudaStatus = hipMalloc((void**)&notkeep_dev_data, dataSize * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }
        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpy(notkeep_dev_data, keep_data, dataSize * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        mapNotKernel << <1, dataSize >> > (notkeep_dev_data, dataSize);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "reduceKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching mapNotKernel!\n", cudaStatus);
            goto Error;
        }
        // Copy output vector from GPU buffer to host memory.
        cudaStatus = hipMemcpy(not_keep_data, notkeep_dev_data, dataSize * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }


        cudaStatus = hipMalloc((void**)&zoffset_dev_data, dataSize * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }
        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpy(zoffset_dev_data, not_keep_data, dataSize * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        // Launch a kernel on the GPU with one thread for each element.
        exscanKernel << <1, dataSize >> > (zoffset_dev_data, dataSize);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "exscanKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching exscanKernel!\n", cudaStatus);
            goto Error;
        }
        // Copy output vector from GPU buffer to host memory.
        cudaStatus = hipMemcpy(zero_offset_data, zoffset_dev_data, dataSize * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }


        // Allocate GPU buffers for three vectors (three input, one output)
        cudaStatus = hipMalloc((void**)&max_dev_data, dataSize * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }
        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpy(max_dev_data, zero_offset_data, dataSize * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        predNonnegReduceKernel << <1, dataSize >> > (max_dev_data, notkeep_dev_data, dataSize);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "predNonnegReduceKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching predNonnegReduceKernel!\n", cudaStatus);
            goto Error;
        }
        cudaStatus = hipMemcpy(max_data, max_dev_data, dataSize * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }


        cudaStatus = hipMalloc((void**)&dev_data, dataSize * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }
        cudaStatus = hipMemcpy(dev_data, data, dataSize * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_new_data, dataSize * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }
        compactMapKernel << <1, dataSize >> > (dev_new_data, dev_data, zoffset_dev_data, max_data[0]+1, ooffset_dev_data, keep_dev_data, dataSize);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "compactMapKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching compactMapKernel!\n", cudaStatus);
            goto Error;
        }

        // Copy output vector from GPU buffer to host memory.
        cudaStatus = hipMemcpy(data, dev_new_data, dataSize * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }
    }

Error:
    hipFree(ooffset_dev_data);
    hipFree(zoffset_dev_data);
    hipFree(max_dev_data);
    hipFree(dev_data);
    hipFree(keep_dev_data);
    hipFree(notkeep_dev_data);
    hipFree(dev_new_data);

    return cudaStatus;
}

